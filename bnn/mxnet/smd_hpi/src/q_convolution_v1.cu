/*!
 * Copyright (c) 2016 by Contributors
 * \file q_convolution.cu
 * \brief Quantized CONV operator
 * \author HPI-DeepLearning
*/

#include "./q_convolution_v1-inl.h"
#include <mshadow/tensor.h>

namespace mshadow {

	inline void QConvolutionV1Forward(int m, int n, int k,
									mxnet::op::xnor_cpu::BINARY_WORD* wmat_binarized,
									Tensor<gpu, 1, float> &workspace,
									const Tensor<gpu, 2, float> &in_col,
									Tensor<gpu, 2, float> &temp_dst) {
		CHECK(false) << "cuda with pre-binarized weights not implemented";
	}

	inline void QConvolutionV1Forward(int m, int n, int k,
									const Tensor<gpu, 2, float> &wmat,
									Tensor<gpu, 1, float> &workspace,
									const Tensor<gpu, 2, float> &in_col,
									Tensor<gpu, 2, float> &temp_dst) {
		//!deprecated! will be removed later
		//cuda::QConvolutionForward(wmat, in_col, temp_dst);
	}

	template<typename DType>
	inline void QConvolutionV1Forward(int m, int n, int k,
									const Tensor<gpu, 2, DType> &wmat,
									Tensor<gpu, 1, DType> &workspace,
									const Tensor<gpu, 2, DType> &in_col,
									Tensor<gpu, 2, DType> &temp_dst) {
		CHECK(false) << "only float supported";
	}

	template<typename DType>
	inline void QConvolutionV1Forward(int m, int n, int k,
									mxnet::op::xnor_cpu::BINARY_WORD* wmat_binarized,
									Tensor<gpu, 1, DType> &workspace,
									const Tensor<gpu, 2, DType> &in_col,
									Tensor<gpu, 2, DType> &temp_dst) {
		CHECK(false) << "only float supported";
	}
} // namespace mshadow

namespace mxnet {
namespace op {

template<>
Operator* CreateOp<gpu>(QConvolutionV1Param param, int dtype,
                        std::vector<TShape> *in_shape,
                        std::vector<TShape> *out_shape,
                        Context ctx) {
  Operator *op = NULL;
  
  MSHADOW_REAL_TYPE_SWITCH(dtype, DType, {
    op = new QConvolutionV1Op<gpu, DType>(param);
  })  	
	return op;
}

}  // namespace op
}  // namespace mxnet

